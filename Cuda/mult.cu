
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <pthread.h>

#define N 1024
#define DOMAIN 100
#define THREADS 4

  int* A_mat;
  int* B_mat;
  int* C_mat;

void print_Mat(int* M);

void* multiply( void* arg );

__global__ void multiply(int* matA, int* matB, int* matC, int size, int Threads){

  //int col= blockIdx.x * blockDim.x + threadIdx.x, i, j, k;
  int i,j,k;
  int row= blockIdx.y * blockDim.y + threadIdx.y;

  int work = N/Threads;

  for(i = row*work; i < row*work+work; ++i){
    for(j = 0; j < N; ++j){
      matC[i*N+j] = 0;
      for(k = 0; k < N; ++k){
        matC[i*N+j] += matA[i*N+k] * matB[k*N+j];
      }
    }
  }
  return;

}


int main(){

  //allocate and populate arrays
  int i, j;
  time_t t;

  //Cuda variables
  int* device_MatA, *device_MatB, *device_MatC;

  //Time variables
  struct timespec start, finish;
  double fin_time;
  srand((unsigned) time(&t));
  
  //allocate and populate arrays with random numbers
  A_mat = (int*)malloc(N * N * sizeof(int));
  B_mat = (int*)malloc(N * N * sizeof(int));
  C_mat = (int*)malloc(N * N * sizeof(int));

  for(i = 0; i  < N; ++i){
    for(j=0; j < N; ++j){
      A_mat[i*N + j] = rand() % DOMAIN +1;
      B_mat[i*N + j] = rand() % DOMAIN +1;
    }
  }

  hipMalloc(&device_MatA, N*N*sizeof(int));
  hipMalloc(&device_MatB, N*N*sizeof(int));
  hipMalloc(&device_MatC, N*N*sizeof(int));
  
  hipMemcpy(device_MatA, A_mat, N*N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_MatB, B_mat, N*N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_MatC, C_mat, N*N*sizeof(int), hipMemcpyHostToDevice);

  clock_gettime(CLOCK_MONOTONIC, &start);

  // -- 1 thread per cell multiply<<<1, dim3(4,4)>>>(deviceA_ptr, deviceB_ptr, deviceC_ptr);
  multiply<<<1, dim3(1, THREADS)>>>(device_MatA, device_MatB, device_MatC, N, THREADS);
  hipDeviceSynchronize();

  hipMemcpy(C_mat, device_MatC, N*N*sizeof(int), hipMemcpyDeviceToHost);

  clock_gettime(CLOCK_MONOTONIC, &finish);
  fin_time = (finish.tv_sec - start.tv_sec);
  fin_time += (finish.tv_nsec - start.tv_nsec) / 1000000000.0;

  //print_Mat(A_mat);
  //print_Mat(B_mat);

  /*
  print_Mat(A_mat);
  printf("\n\n");
  print_Mat(B_mat);
  printf("\n\n");
  print_Mat(C_mat);
*/
  printf("Time to complete: %f\n", fin_time);

  // Multiply arrays
  free(A_mat);
  free(B_mat);
  free(C_mat);
  
  hipFree(device_MatA);
  hipFree(device_MatB);
  hipFree(device_MatC);
  return 0;
}


void print_Mat(int* M){

  int i, j;
  for(i = 0; i < N; ++i){
    for(j = 0; j < N; ++j){
      printf("%d ", M[i*N+j]);
    }
    printf("\n");
  }
}

void* multiply( void* arg ){
  
  int work = N/THREADS; //count iterations

  int* thread_id = (int *)arg;
  

  int i,j,k;
  for(i = (*thread_id)*work; i < (*thread_id)*work + work; ++i){
    for(j = 0; j < N; ++j){
      C_mat[i*N + j] = 0;
      for(k = 0; k < N; ++k){
        C_mat[i*N + j] += A_mat[i*N + k] * B_mat[k*N + j];
      }
    }
  }

  thread_id = NULL;
  free(arg);
  arg = NULL;

  return NULL;

}

